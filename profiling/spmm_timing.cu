#include <cstdio>             
#include <cstdlib>            
#include <sparsify.me/ampere.hxx>

#define CHECK_CUSPARSE(func)                                                   
{                                                                              
    hipsparseStatus_t status = (func);                                          
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",        
               __LINE__, hipsparseGetErrorString(status), status);              
        return EXIT_FAILURE;                                                   
    }                                                                          
}

int main(int argc, char** argv) {
  using namespace sparsifyme;
  using type_t = float;

  if(argc != 5) {
    throw "Invalid # of args. Usage: ./spmm_timing m n k b";
  }

  int major_cc, minor_cc;
  hipDeviceGetAttribute(&major_cc, hipDeviceAttributeComputeCapabilityMajor, 0);
  hipDeviceGetAttribute(&minor_cc, hipDeviceAttributeComputeCapabilityMinor, 0);
  if (!(major_cc == 8 && minor_cc == 0)) {
      std::printf("\ncusparseLt is supported only on GPU devices with"
                  " compute capability == 8.0, current: %d.%d\n\n",
                   major_cc, minor_cc);
      return EXIT_UNSUPPORTED;
  }

  hipDataType valueType;

  std::size_t m = std::stoi(argv[2]);
  std::size_t n = std::stoi(argv[3]);
  std::size_t k = std::stoi(argv[4]);


  // Initialize host pointers
  type_t* hA = malloc(m * k * sizeof(type_t));
  type_t* hB = malloc(k * n * sizeof(type_t));
  type_t* hC = malloc(m * n * sizeof(type_t));
  for(int i = 0; i < m * k; i++) {
    hA[i] = static_cast<type_t>(static_cast<float>(std::rand() % 100));
  }
  for(int i = 0; i < m * k; i++) {
    hB[i] = static_cast<type_t>(static_cast<float>(std::rand() % 100));
  }
  float alpha = 1.0f;
  float beta = 0.f;

  // Allocate device pointers and move data to GPU from host pointers
  type_t* dA, dB, dC;
  hipMalloc((**void)&dA, m*k*sizeof(type_t));
  hipMalloc((**void)&dB, k*n*sizeof(type_t));
  hipMalloc((**void)&dC, m*n*sizeof(type_t));

  hipMemcpy(dA,hA,m*k*sizeof(type_t), hipMemcpyHostToDevice);
  hipMemcpy(dB,hB,k*n*sizeof(type_t), hipMemcpyHostToDevice);
  hipMemcpy(dC,hC,m*n*sizeof(type_t), hipMemcpyHostToDevice);

  // Call Ampere Functions
  auto times = sparsifyme::ampere_spmm(&dA, &dB, &dC, m,n,k);
  // std::cout << "Matrix Sizes (m, n, k) = (" << m << ", " << n << ", " << k << ")" << std::endl;
  std::cout << times[0] << ", " << times[1] << ", " << times[2] << std::endl;

}